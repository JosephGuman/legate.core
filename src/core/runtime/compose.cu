#include "hip/hip_runtime.h"
#include "legion.h"

#include <memory>

#include "core/runtime/resource.h"
#include "core/task/exception.h"
#include "core/utilities/typedefs.h"
#include "default_mapper.h"
#include "legion/legion_mapping.h"
#include "legion/legion_c.h"

#include "compose.h"

namespace Legion{

    using namespace Legion;
    using namespace Legion::Mapping;


    __global__
    void myKernel(int r0l, int r0h, int r1l, int r1h, FieldAccessor<READ_WRITE,double,2,coord_t,Realm::AffineAccessor<double,2,coord_t> > writeTo){
        int x = blockIdx.x * blockDim.x + threadIdx.x + r0l;
        int y = blockIdx.y * blockDim.y + threadIdx.y + r1l;

        if(x <= r0h && y <= r1h){
            Point<2> p = Point<2>(x,y);

            writeTo[p] = writeTo[p] * 2;
        }
    }
    
    __host__
    void fake_task(
        const Legion::Task *task, 
        const std::vector<Legion::PhysicalRegion> &regions,
        Legion::Context ctx, Legion::Runtime *runtime
    )
    {
        Rect<2> rect = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
        
        FieldAccessor<READ_WRITE,double,2,coord_t,Realm::AffineAccessor<double,2,coord_t> > writeTo(regions[0],1000);

        dim3 threads_per_block(16,16);
        dim3 num_blocks((rect.hi[0] - rect.lo[0] + threads_per_block.x) / threads_per_block.x, (rect.hi[1] - rect.lo[1] + threads_per_block.y) / threads_per_block.y);


        myKernel<<<num_blocks,threads_per_block>>>(rect.lo[0], rect.hi[0], rect.lo[1], rect.hi[1], writeTo);
    }
    
    __host__
    int ComposeClassC::registerFake(){
        Legion::Runtime* runtime = Legion::Runtime::get_runtime();
        
        int printOp = runtime->generate_dynamic_task_id();
        {
            TaskVariantRegistrar registrar(printOp, "designed_to_compose");
            registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
            runtime->register_task_variant<fake_task>(registrar);
        }

        return printOp;
    }
}